#include <iostream>
#include <math.h>
#include <cassert>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printMatrix(int m, int n, const double *A, int lda, const char *name)
{
    for (int row=0; row<m; row++) {
        for (int col=0; col<n; col++) {
            double Areg = A[row + (col*lda)];
            std::cout << name << "(" << row+1 << "," << col+1 << ") = " << Areg << std::endl;
        }
    }
}

int main(int argc, char **argv)
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;

    const int m = 3;
    const int lda = m;
    const int ldb = m;

    double A[lda*m] = {1.0 , 4.0, 7.0, 2.0, 5.0, 8.0, 3.0, 6.0, 10.0};
    double B[m] = {1.0, 2.0, 3.0};
    double X[m];
    double LU[lda*m];

    int Ipiv[m];
    int info = 0;

    double *d_A = NULL;
    double *d_B = NULL;
    int *d_Ipiv = NULL;
    int *d_info = NULL;
    int lwork = 0;
    double *d_work = NULL;

    const int pivot_on = false;

    std::cout << "example of getrf" << std::endl;

    if (pivot_on) {
        std::cout << "pivot in on : compute P*A = L*U" << std::endl;
    }

    else {
        std::cout <<  "pivot is off : compute A = L*U" << std::endl;
    }

    std::cout << "A = (matlab base-1)" << std::endl;
    printMatrix(m, m, A, lda, "A");
    std::cout << "======================================================" << std::endl;

    std::cout << "B = (matlab base-1)" << std::endl;
    printMatrix(m, 1, B, ldb, "B");
    std::cout << "======================================================" << std::endl;

    //step 1 : create cuSolver handle, bind a stream
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    //step 2 : copy A to device
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(double) * m);
    cudaStat3 = hipMalloc ((void**)&d_Ipiv, sizeof(int) * m);
    cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*lda*m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double)*m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    //step 3 : query working space of getrf
    status = hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    //step 4 : LU factorisation
    if (pivot_on) {
        status = hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, d_Ipiv, d_info);
    }

    else {
        status = hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, NULL, d_info);
    }
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    if (pivot_on) {
        cudaStat1 = hipMemcpy(Ipiv, d_Ipiv, sizeof(int)*m, hipMemcpyDeviceToHost);
    }
    cudaStat2 = hipMemcpy(LU, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    if (0 > info) {
        std::cout << -info << "-th parameter is wrong" << std::endl;
        exit(1);
    }
    if (pivot_on) {
        std::cout << "pivoting sequence, matlab base-1" << std::endl;
        for (int j=0; j<m; j++) {
            std::cout << "Ipiv(" << j+1 << ") = " << Ipiv[j] << std::endl;
        }
    }
    std::cout << "L and U = (matlab base-1)" << std::endl;
    printMatrix(m, m, LU, lda, "LU");
    std::cout << "======================================================" << std::endl;

    //step 5 : solve A*X=B
    if (pivot_on) {
        status = hipsolverDnDgetrs(
                            cusolverH, 
                            HIPBLAS_OP_N, 
                            m,
                            1,
                            d_A,
                            lda,
                            d_Ipiv,
                            d_B,
                            ldb,
                            d_info);
    }else {
        status = hipsolverDnDgetrs(
                            cusolverH,
                            HIPBLAS_OP_N,
                            m,
                            1,
                            d_A,
                            lda,
                            NULL,
                            d_B,
                            ldb,
                            d_info);
    }
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(X, d_B, sizeof(double)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    std::cout << "X = (matlab base-1)" << std::endl;
    printMatrix(m, 1, X, ldb, "X");
    std::cout << "======================================================" << std::endl;

    //free resources
    if (d_A     ) hipFree(d_A);
    if (d_B     ) hipFree(d_B);
    if (d_Ipiv  ) hipFree(d_Ipiv);
    if (d_info  ) hipFree(d_info);
    if (d_work  ) hipFree(d_work);

    if (cusolverH   ) hipsolverDnDestroy(cusolverH);
    if (stream      ) hipStreamDestroy(stream);

    hipDeviceReset();

    return 0;
}